#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _MAX_UNPOOL_EXPORT
#define _MAX_UNPOOL_EXPORT 
#endif
 
#include <iostream>
#include <assert.h>
#include <nvmatrix_kernels.cuh>
#include <nvmatrix.cuh>
#include <conv_util.cuh>

using namespace std;


template<int B_Y, int B_X, int imgsPerThread, int filtersPerThread, bool add, bool checkCaseBounds>
__global__ void kLocalMaxInverse(float* imgs, float* maxGrads, float* maxActs, float* target, const int imgSize, const int numFilters,
                              const int numImages, const int subsX, const int startX, const int strideX, const int outputsX,
                              const float scaleTargets, const float scaleOutputs) {
    __shared__ float shImgs[B_Y*filtersPerThread][B_X*imgsPerThread];
    const int numImgBlocks = DIVUP(numImages,B_X*imgsPerThread);
    const int blockPxX = blockIdx.x / numImgBlocks;
    const int blockPxY = blockIdx.y / (numFilters/(B_Y*filtersPerThread));
    
    const int blockImgIdx = (blockIdx.x % numImgBlocks) * B_X * imgsPerThread;
    const int blockFilterIdx = (blockIdx.y % (numFilters/(B_Y*filtersPerThread))) * B_Y * filtersPerThread;
    
    const int blockPx = blockPxY * imgSize + blockPxX;
    const int numOutputs = outputsX * outputsX;
    const int imgPixels = imgSize * imgSize;

    const int startOutputY = blockPxY - startX < subsX ? 0 : 1 + (blockPxY - startX - subsX) / strideX;
    const int endOutputY = MIN(outputsX, 1 + (blockPxY - startX) / strideX);
    const int startOutputX = blockPxX - startX < subsX ? 0 : 1 + (blockPxX - startX - subsX) / strideX;
    const int endOutputX = MIN(outputsX, 1 + (blockPxX - startX) / strideX);
    
    const int imgIdx = blockImgIdx + threadIdx.x;
    
    imgs += ((blockFilterIdx + threadIdx.y) * imgPixels + blockPx) * numImages + imgIdx;
    maxGrads += ((blockFilterIdx + threadIdx.y) * numOutputs) * numImages 
            + imgIdx;
    maxActs += ((blockFilterIdx + threadIdx.y) * numOutputs) * numImages 
            + imgIdx;
    
    target += ((blockFilterIdx + threadIdx.y) * imgPixels + blockPx) * numImages + imgIdx;
    
    float prod[filtersPerThread][imgsPerThread];
    #pragma unroll
    for (int f = 0; f < filtersPerThread; f++) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            prod[f][i] = 0;
        }
    }
    
    if  (blockPxX >= startX && blockPxX < startX + strideX * (outputsX-1) + subsX 
         && blockPxY >= startX && blockPxY < startX + strideX * (outputsX-1) + subsX) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || imgIdx + i * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    shImgs[threadIdx.y + B_Y * f][threadIdx.x + B_X * i] = imgs[f * B_Y * imgPixels * numImages + i * B_X];
                }
            }
        }
        for (int my = startOutputY; my < endOutputY; my++) {
            for (int mx = startOutputX; mx < endOutputX; mx++) {
                const int outputIdx = my * outputsX + mx;
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    if (!checkCaseBounds || imgIdx + i * B_X < numImages) {
                        #pragma unroll
                        for (int f = 0; f < filtersPerThread; f++) {
                            const float ma = maxActs[(f * B_Y * numOutputs + outputIdx) * numImages + i * B_X]; 
                            const float mg = maxGrads[(f * B_Y * numOutputs + outputIdx) * numImages + i * B_X];
                            const float img = shImgs[threadIdx.y + B_Y * f][threadIdx.x + B_X * i];
			    
			    if (img == ma) {
				prod[f][i] = mg;
			    }
                        }
                    }
                }
            }
        }
    }
    if (!add) {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || imgIdx + i * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    target[f * B_Y * imgPixels * numImages + i * B_X] = prod[f][i];
                }
            }
        }
    } else {
        #pragma unroll
        for (int i = 0; i < imgsPerThread; i++) {
            if (!checkCaseBounds || imgIdx + i * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    target[f * B_Y * imgPixels * numImages + i * B_X] = scaleTargets * target[f * B_Y * imgPixels * numImages + i * B_X] + scaleOutputs * prod[f][i];
                }
            }
        }
    }
}

void convLocalMaxInverse(NVMatrix& images, NVMatrix& maxGrads, NVMatrix& maxActs, NVMatrix& target,
                      int subsX, int startX, int strideX, int outputsX) {
    convLocalMaxInverse(images, maxGrads, maxActs, target, subsX, startX, strideX, outputsX, 0, 1);
}

/*
 * imgs:        (numFilters * imgPixels, numImages)
 * maxGrads:    (numFilters * numOutputs, numImages)
 * maxActs:    (numFilters * numOutputs, numImages)
 * target:      (numFilters * imgPixels, numImages)
 */
void convLocalMaxInverse(NVMatrix& images, NVMatrix& maxGrads, NVMatrix& maxActs, NVMatrix& target,
                      int subsX, int startX, int strideX, int outputsX, float scaleTargets, float scaleOutput) {
    int outputs = outputsX * outputsX;
    int numImages = images.getNumCols();
    int numFilters = maxGrads.getNumRows() / outputs;
    int imgPixels = images.getNumRows() / numFilters;
    assert(images.getNumRows() == numFilters * imgPixels);
    int imgSize = int(sqrt((double)imgPixels));
    
    assert(imgSize * imgSize == imgPixels);
    assert(maxGrads.getNumRows() == numFilters * outputs);
    assert(maxGrads.getNumCols() == numImages);
    assert(!images.isTrans());
    assert(!target.isTrans());
    assert(!maxGrads.isTrans());
    assert(!maxActs.isTrans());
    assert(images.isContiguous());
    assert(maxGrads.isContiguous());
    assert(maxActs.isContiguous());
    assert(maxGrads.isSameDims(maxActs));
    assert(numFilters % 16 == 0);
//    assert(numImages % 128 == 0);
    
    assert(strideX <= subsX);
    
    target.resize(images);
    assert(target.isContiguous());
    int imgsPerThread = numImages % 128 == 0 ? 4 : numImages % 64 == 0 ? 2 : 1;
    int checkCaseBounds = numImages % (32*imgsPerThread) != 0;
    dim3 threads(32, 4);
    dim3 blocks(DIVUP(numImages,32*imgsPerThread) * imgSize, (numFilters / (4 * 2)) * imgSize);
    
    if (imgsPerThread == 4) {
        if  (checkCaseBounds) {
            if (scaleTargets == 0 && scaleOutput == 1) {
                kLocalMaxUndo<4, 32, 4, 2, false, true><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            } else {
                kLocalMaxInverse<4, 32, 4, 2, true, true><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            }
        } else {
            if (scaleTargets == 0 && scaleOutput == 1) {
                kLocalMaxInverse<4, 32, 4, 2, false, false><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            } else {
                kLocalMaxInverse<4, 32, 4, 2, true, false><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            }
        }
    } else if (imgsPerThread == 2) {
        if  (checkCaseBounds) {
            if (scaleTargets == 0 && scaleOutput == 1) {
                kLocalMaxInverse<4, 32, 2, 2, false, true><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            } else {
                kLocalMaxInverse<4, 32, 2, 2, true, true><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            }
        } else {
            if (scaleTargets == 0 && scaleOutput == 1) {
                kLocalMaxInverse<4, 32, 2, 2, false, false><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            } else {
                kLocalMaxInverse<4, 32, 2, 2, true, false><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            }
        }
    } else {
        if  (checkCaseBounds) {
            if (scaleTargets == 0 && scaleOutput == 1) {
                kLocalMaxInverse<4, 32, 1, 2, false, true><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            } else {
                kLocalMaxInverse<4, 32, 1, 2, true, true><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            }
        } else {
            if (scaleTargets == 0 && scaleOutput == 1) {
                kLocalMaxInverse<4, 32, 1, 2, false, false><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            } else {
                kLocalMaxInverse<4, 32, 1, 2, true, false><<<blocks, threads>>>(images.getDevData(), maxGrads.getDevData(), maxActs.getDevData(), target.getDevData(),
                                                                imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
            }
        }
    }

    cutilCheckMsg("convLocalMaxUndo: kernel execution failed");
}
